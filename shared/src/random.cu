#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>
#include <time.h>

#include "random.cuh"

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

/**
 * Fill an array with random floats using the CURAND function.
 * \param devData The chunk of memory you want to fill with floats within the range (0,1]
 * \param n The size of the chunk of data
 */
int randFloats(float *&devData, const size_t n) {
    // The generator, used for random numbers
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    
    // Set seed to be the current time (note that calls close together will have same seed!)
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

    // Generate n floats on device
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

    // Cleanup
    CURAND_CALL(hiprandDestroyGenerator(gen));
    return EXIT_SUCCESS;
}
